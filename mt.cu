#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>
#include <random>
#include <hip/hip_vector_types.h>



struct DD {

  int *m_xs;
  double *m_ps;
  size_t m_size;

  DD(int *xs, double *ps, size_t size) {
    this->m_xs = xs;
    this->m_ps = ps;
    this->m_size = size;
  }

  void print() {
    std::cout << "Size: " << m_size << std::endl;
    for (int i = 0; i < m_size; i++) {
      std::cout << m_xs[i] << " " << m_ps[i] << std::endl;
    }
  }
};


void print_array(float *ar, int len){

  for (int i = 0; i < len; i++){
    std::cout << "ar @ pos " << i << " = " << ar[i] << "\n";
  }
}

__global__ void convolution_kernel(float *p1, float *p2, float *p3, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y* blockDim.y + threadIdx.y;
  
  //p3[i] = p1[i] + p2[i];
  //if ((i + j) < size*size){
  if (i  < size && j < size){

    p3[i * size + j] = p1[i] * p2[j];
  }

  //int i = threadIdx.x;
  //int j = threadIdx.y;
  //if (threadIdx.x <= size &&  threadIdx.y <= size) {
  //if (i <= size &&  j <= size) {
    //p3[threadIdx.x * size + threadIdx.y] = p1[threadIdx.x] * p2[threadIdx.y];
  //p3[i * size + j] = p1[i] * p2[j];
  //p3[threadIdx.x * size + threadIdx.y] = p1[threadIdx.x] * p2[threadIdx.y];
  //	}
}


__global__ void add_kernel(float *p1, float *p2, float *p3, int size){
  int i = threadIdx.x;
  p3[i] = p1[i] + p2[i];
}

DD convolution(float *p1, float *p2, int size) {
  std::cout << "Convolution" << std::endl;
  return DD(NULL, NULL, 0);
}


void populate_with_randvalues(float *ar, int len){
  float sum = 0.0;
  std::mt19937 r_gen{std::random_device{}()};
  std::uniform_real_distribution<double> dist(0.0, 1.0);

  for (int i = 0; i < len; i++) {
    ar[i] = (float) dist(r_gen);
    sum += ar[i];
  }
  for (int i = 0; i < len; i++) {
    ar[i] = ar[i] / sum;
  }
  std::cout << "Sum: " << sum << std::endl;
}


template <typename T>
int check_alloc(T *ptr){
  if (ptr == NULL){
    std::cout << "Error allocating memory\n";
    return -1;
	}
  else{
    std::cout << "Memory allocated successfully.\n";
    return 1;
  }
}


void manual_convolution(float* p1, float* p2, float*p3, int len){
  for (long i = 0; i<len; i++){
    for (long j = 0; j<len; j++){
      p3[i * len + j] = p1[i] * p2[j];
  }
  }
}


int is_equal(float *p1, float *p2, long len){

  std::cout << "is_equal called\n";

  long wrong = 0;
  for (long i = 0; i < len; i++){
    if (std::abs(p1[i] - p2[i]) > 1e-6){
      wrong += 1;
    }
    else{
      continue;
    }
  }
  std::cout << "is_equal comparison done\n";

  if (wrong == 0) {

    return 1;
  }
  else{
    std::cout << "Wrong values at " << wrong <<" positions: ";
    return 0;
	}
}

int main() {

  std::cout << "size of int: " << sizeof(int) << "\n";
  std::cout << "size of long: " << sizeof(long) << "\n";
  


  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  std::cout << "Device count: " << deviceCount << std::endl;



  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  std::cout << "Device name: " << prop.name << std::endl;
  std::cout << "Memory: " << prop.totalGlobalMem << std::endl;
  std::cout << "Shared memory per block: " << prop.sharedMemPerBlock << std::endl;
  std::cout << "Registers per block: " << prop.regsPerBlock << std::endl;
  std::cout << "Warp size: " << prop.warpSize << std::endl;
  std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
  std::cout << "Max threads dim: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << std::endl;

  size_t SIZE = 100000;
  size_t inp_size_alloc = SIZE * sizeof(float);
  size_t res_size_alloc = SIZE * SIZE * sizeof(float);

  std::mt19937 r_gen{};
  
  std::uniform_real_distribution<double> dist(0.0, 1.0);
  for (int i = 0; i < 10; i++) {
		std::cout << dist(r_gen) << std::endl;
	}


  //int xs[] = {1, 2, 3, 4, 5};
  //double ps[] = {.2, .2, .2, .2, .2};
  //DD d1(xs, ps, 5);
  //DD d2(xs, ps, 5);


  float *p1 = (float *) malloc(inp_size_alloc);
  float *p2 = (float *) malloc(inp_size_alloc);
  //float p1[SIZE] = {0.0};
  check_alloc<float>(p1);
  //float p2[SIZE] = {0.0};
  check_alloc<float>(p2);
  float *p3 = (float *) malloc(res_size_alloc);
  check_alloc<float>(p3);
  float *p3_manual = (float *) malloc(res_size_alloc);
  check_alloc<float>(p3_manual);


  populate_with_randvalues(p1, SIZE);
  populate_with_randvalues(p2, SIZE);
  //print_array(p1, SIZE);
  //print_array(p2, SIZE);

  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);
  std::cout << "Free memory: " << free_mem << std::endl;
  std::cout << "Total memory: " << total_mem << std::endl;

  float *d_p1;
  hipMalloc(&d_p1, inp_size_alloc);
  std::cout << "Allocated memory on device for p1\n";


  float *d_p2;
  hipMalloc(&d_p2, inp_size_alloc);
  std::cout << "Allocated memory on device for p2\n";

  float *d_p3; 
  hipMalloc(&d_p3, res_size_alloc);

  float *d_p4; 
  hipMalloc(&d_p4, inp_size_alloc);

  std::cout << "Allocated memory on device for p3\n";

  hipMemcpy(d_p1, p1, inp_size_alloc, hipMemcpyHostToDevice);
  hipMemcpy(d_p2, p2, inp_size_alloc, hipMemcpyHostToDevice);
  //cudaMemcpy(d_p3, p3, res_size_alloc, cudaMemcpyHostToDevice);



  //int num_threads = 512;
  int num_threads = 128;
  int num_blocks = 5;
  //int num_blocks = std::ceil(SIZE / num_threads);
  int num_blocks_x = (SIZE + num_threads - 1) / num_threads;
  int num_blocks_y = (SIZE + num_threads - 1) / num_threads;
  dim3 ts(num_threads, num_threads, 1);
  dim3 bs(num_blocks_x, num_blocks_y, 1);
  std::cout << "Num blocks: " << num_blocks_x << std::endl;
  std::cout << "Num threads: " << num_threads << std::endl;
  
  convolution_kernel<<<bs, ts>>>(d_p1, d_p2, d_p3, SIZE);
  //convolution_kernel<<<1, num_threads>>>(d_p1, d_p2, d_p3, SIZE);
  std::cout << "Kernel executed\n";

  //convolution_kernel<<<bs, ts>>>(d_p1, d_p2, d_p3, num_threads);
  
  // print_array(p3, 10);
  hipMemcpy(p3, d_p3, res_size_alloc, hipMemcpyDeviceToHost);
  //print_array(p3, SIZE*SIZE);


  //manual_convolution(p1, p2, p3_manual, SIZE);
  //std::cout << "Manual convolution exectuted\n";
  //print_array(p3_manual, SIZE*SIZE);

  //if(is_equal(p3, p3_manual, SIZE*SIZE) == 0) std::cout << "Not Equal\n"

  //float *p4 = (float *) malloc(SIZE * sizeof(float));
  //*p4 = 0.0;
  //add_kernel<<<1, num_threads>>>(d_p1, d_p2, d_p4, num_threads);
  //cudaMemcpy(p4, d_p4, inp_size_alloc, cudaMemcpyDeviceToHost);
  //std::cout << "Finished\n";

}
